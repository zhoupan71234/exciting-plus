#include "gpu_interface.h"
#include "linalg_gpu.h"

extern "C" void gpu_zgemms(int transa, int transb, int32_t m, int32_t n, int32_t k, 
                           complex16 alpha, complex16 *a, int32_t lda, complex16 *b, 
                           int32_t ldb, complex16 beta, complex16 *c, int32_t ldc)
{
    assert(sizeof(hipDoubleComplex) == sizeof(complex16));
    
    const hipblasOperation_t trans[] = {HIPBLAS_OP_N, HIPBLAS_OP_T, HIPBLAS_OP_C};

    if (hipblasZgemm(cublas_handle(), trans[transa], trans[transb], m, n, k, (hipDoubleComplex *)(&alpha), (hipDoubleComplex *)a, lda, 
                    (hipDoubleComplex *)b, ldb, (hipDoubleComplex *)(&beta), (hipDoubleComplex *)c, ldc) != HIPBLAS_STATUS_SUCCESS)
    {
        printf("failed to execute hipblasZgemm() \n");
        exit(0);
    }
}

extern "C" void gpu_zhegvx(int32_t n, int32_t nv, double abstol, void *a, void *b,
                           double *eval, void *z, int32_t ldz)
{
    magma_int_t m1, info;

    magma_int_t nb = magma_get_zhetrd_nb(n);
    magma_int_t lwork = 2 * n * (nb + 1);
    magma_int_t lrwork = 7 * n;
    magma_int_t liwork = 6 * n;
    
    hipDoubleComplex *h_work;
    double *rwork, *w1;
    magma_int_t *iwork, *ifail;
    
    w1 = (double *)malloc(n * sizeof(double));
    h_work = (hipDoubleComplex *)malloc(lwork * sizeof(hipDoubleComplex));
    rwork = (double *)malloc(lrwork * sizeof(double));
    iwork = (magma_int_t *)malloc(liwork * sizeof(magma_int_t));
    ifail = iwork + 5 * n;

    magma_zhegvx(1, 'V', 'I', 'U', n, (hipDoubleComplex *)a, n, (hipDoubleComplex *)b, n, 0.0, 0.0, 1, nv, abstol, 
                 &m1, w1, (hipDoubleComplex *)z, ldz, h_work, lwork, rwork, iwork, ifail, &info);

    memcpy(eval, &w1[0], nv * sizeof(double)); 
    
    free(iwork);
    free(rwork);
    free(w1);
    free(h_work);
}
 
