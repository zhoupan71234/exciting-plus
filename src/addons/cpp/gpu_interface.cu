#include "gpu_interface.h"

hipDeviceProp_t& cuda_devprop()
{
    static hipDeviceProp_t devprop;

    return devprop;

}

hipblasHandle_t& cublas_handle()
{
    static hipblasHandle_t handle;
    static bool init = false;

    if (!init)
    {
        if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
        {
            printf("hipblasCreate() failed \n");
            exit(0);
        }
        init = true;
    }
    
    return handle;
}

extern "C" void init_gpu()
{
    int count;
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("init_gpu: failed to execute hipGetDeviceCount() \n");
        return;
    }

    if (count == 0)
    {
        printf("init_gpu: no avaiable devices\n");
    }

    hipDeviceProp_t devprop;
     
    if (hipGetDeviceProperties(&devprop, 0) != hipSuccess)
    {
        printf("init_gpu: failed to execute hipGetDeviceProperties()\n");
        return;
    }
    
    printf("name                        : %s \n", devprop.name);
    printf("major                       : %i \n", devprop.major);
    printf("minor                       : %i \n", devprop.minor);
    printf("asyncEngineCount            : %i \n", devprop.asyncEngineCount);
    printf("canMapHostMemory            : %i \n", devprop.canMapHostMemory);
    printf("clockRate                   : %i kHz \n", devprop.clockRate);
    printf("concurrentKernels           : %i \n", devprop.concurrentKernels);
    printf("ECCEnabled                  : %i \n", devprop.ECCEnabled);
    printf("l2CacheSize                 : %i kB \n", devprop.l2CacheSize/1024);
    printf("maxGridSize                 : %i %i %i \n", devprop.maxGridSize[0], devprop.maxGridSize[1], devprop.maxGridSize[2]);
    printf("maxThreadsDim               : %i %i %i \n", devprop.maxThreadsDim[0], devprop.maxThreadsDim[1], devprop.maxThreadsDim[2]);
    printf("maxThreadsPerBlock          : %i \n", devprop.maxThreadsPerBlock);
    printf("maxThreadsPerMultiProcessor : %i \n", devprop.maxThreadsPerMultiProcessor);
    printf("memoryBusWidth              : %i bits \n", devprop.memoryBusWidth);
    printf("memoryClockRate             : %i kHz \n", devprop.memoryClockRate);
    printf("memPitch                    : %i \n", devprop.memPitch);
    printf("multiProcessorCount         : %i \n", devprop.multiProcessorCount);
    printf("regsPerBlock                : %i \n", devprop.regsPerBlock);
    printf("sharedMemPerBlock           : %i kB \n", devprop.sharedMemPerBlock/1024);
    printf("totalConstMem               : %i kB \n", devprop.totalConstMem/1024);
    printf("totalGlobalMem              : %i kB \n", devprop.totalGlobalMem/1024);
}

extern "C" void gpu_malloc(void **ptr, int size)
{
    if (hipMalloc(ptr, size) != hipSuccess)
    {
        printf("failed to execute hipMalloc() \n");
        exit(0);
    }
}

extern "C" void gpu_free(void *ptr)
{
    if (hipFree(ptr) != hipSuccess)
    {
        printf("failed to execute hipFree() \n");
        exit(0);
    }
}

extern "C" void gpu_copy_to_device(void *target, void *source, int size)
{
    if (hipMemcpy(target, source, size, hipMemcpyHostToDevice) != hipSuccess)
    {
        printf("failed to execute hipMemcpy(hipMemcpyHostToDevice)\n");
        exit(0);
    }
}

extern "C" void gpu_copy_to_host(void *target, void *source, int size)
{
    if (hipMemcpy(target, source, size, hipMemcpyDeviceToHost) != hipSuccess)
    {
        printf("failed to execute hipMemcpy(hipMemcpyDeviceToHost)\n");
        exit(0);
    }
}

extern "C" void gpu_mem_zero(void *ptr, int size)
{
    if (hipMemset(ptr, 0, size) != hipSuccess)
    {
        printf("failed to execute hipMemset()\n");
        exit(0);
    }
}

